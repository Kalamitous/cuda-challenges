#include <hip/hip_runtime.h>

__global__ void invert_kernel(unsigned char* image, int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < width * height) {
        image[i * 4] = 255 - image[i * 4];
        image[i * 4 + 1] = 255 - image[i * 4 + 1];
        image[i * 4 + 2] = 255 - image[i * 4 + 2];
    }
}
// image_input, image_output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(unsigned char* image, int width, int height) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;

    invert_kernel<<<blocksPerGrid, threadsPerBlock>>>(image, width, height);
    hipDeviceSynchronize();
}