#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int strideX = blockDim.x * gridDim.x;
    int strideY = blockDim.y * gridDim.y;

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = row; i < M; i += strideY) {
        for (int k = col; k < K; k += strideX) {
            float sum = 0.0f;
            for (int j = 0; j < N; j++) {
                sum += A[i * N + j] * B[j * K + k];
            }
            C[i * K + k] = sum;
        }
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* A, const float* B, float* C, int M, int N, int K) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
