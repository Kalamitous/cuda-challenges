#include <hip/hip_runtime.h>

// reference: https://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm

__global__ void bitonic_sort_step(float* data, int j, int k) {
    // shared memory is not useful here since each thread accesses unique elements in data

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int ixj = i ^ j; // the index j distance away from i
    // xor because we want pairwise partners
    // i.e. if i is 1 and we determine that its partner is 2,
    // then when i is 2, its partner must be 1

    // using the above example, avoids duplicate comparison (when i is 2)
    if (ixj > i) {
        float a = data[i];
        float b = data[ixj];
        if ((i & k) == 0) { // == has higher precedence than &
            // sort ascending
            if (a > b) {
                data[i] = b;
                data[ixj] = a;
            }
        } else {
            // sort descending
            if (a < b) {
                data[i] = b;
                data[ixj] = a;
            }
        }
    }
}

// data is device pointer
extern "C" void solve(float* data, int N) {
    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    // k selects the bit pos that determines ascending/descending exchange
    // k is also the length of the subsequences we are sorting
    // k <= N because the final iteration merges the two halves of the full sequence
    for (int k = 2; k <= N; k *= 2) {
        // j is the distance between the elements we compare
        for (int j = k >> 1; j > 0; j >>= 1) {
            bitonic_sort_step<<<threads, blocks>>>(data, j, k);
        }
    }
}